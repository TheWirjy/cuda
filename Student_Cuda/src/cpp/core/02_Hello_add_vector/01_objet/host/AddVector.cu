#include "hip/hip_runtime.h"
#include "AddVector.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVector::AddVector(float* ptrV1, float* ptrV2, float* ptrW, int n) :
	ptrV1(ptrV1), ptrV2(ptrV2), ptrW(ptrW), n(n)
    {
    this->sizeOctet = n * sizeof(float); // octet

    // MM
	{
	// MM (malloc Device)
	    {
	    HANDLE_ERROR(hipMalloc(&ptrDevV1, sizeOctet));
	    HANDLE_ERROR(hipMalloc(&ptrDevV2, sizeOctet));
	    HANDLE_ERROR(hipMalloc(&ptrDevW, sizeOctet));
	    }

	// MM (memset Device)
	    {
	    HANDLE_ERROR(hipMemset(ptrDevW, 0, sizeOctet));
	    }

	// MM (copy Host->Device)
	    {
	    HANDLE_ERROR(hipMemcpy(ptrDevV1, ptrV1, sizeOctet, hipMemcpyHostToDevice));
	    HANDLE_ERROR(hipMemcpy(ptrDevV2, ptrV2, sizeOctet, hipMemcpyHostToDevice));
	    }

	Device::lastCudaError("AddVector MM (end allocation)"); // temp debug
	}

    // Grid
	{
	this->dg = dim3(16, 2, 1); // disons, a optimiser selon le gpu
	this->db = dim3(32, 4, 1); // disons, a optimiser selon le gpu

	Device::gridHeuristic(dg, db);
	}
    }

AddVector::~AddVector(void)
    {
    //MM (device free)
	{
	HANDLE_ERROR(hipFree(ptrDevV1));
	HANDLE_ERROR(hipFree(ptrDevV2));
	HANDLE_ERROR(hipFree(ptrDevW));

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void AddVector::run()
    {
    Device::lastCudaError("addVecteur (before)"); // temp debug
    addVector<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n); // assynchrone
    Device::lastCudaError("addVecteur (after)"); // temp debug

    Device::synchronize(); // Temp, only for printf in  GPU

    // MM (Device -> Host)
	{
	HANDLE_ERROR(hipMemcpy(ptrW, ptrDevW, sizeOctet, hipMemcpyDeviceToHost)); // barriere synchronisation implicite
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
